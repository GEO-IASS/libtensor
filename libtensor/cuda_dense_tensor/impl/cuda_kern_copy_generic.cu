#include "hip/hip_runtime.h"
#include "cuda_kern_copy_generic.h"
#include "cuda_kern_copy_2d.h"
#include "cuda_kern_copy_4d.h"
#include "cuda_kern_copy_6d.h"
#include <iostream>

namespace libtensor {

//template<size_t N>
const char *cuda_kern_copy_generic::k_clazz = "cuda_kern_copy_generic";

//generic implementation of copy operation doesn't do permutation!
//template<size_t N>
void cuda_kern_copy_generic::run() {

	//no default copy
	//do nothing
}


template<size_t N>
cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<N> &dimsa,
		const permutation<N> &perma, const double &c, const double &d) {

	permutation<N> perma_i(perma, true);
	dimensions<N> dimsb(dimsa);
	dimsb.permute(perma);

	sequence<N, size_t>  map(0);
		for (size_t i = 0; i < N ; i++) {
		map[i] = i;
	}
	perma_i.apply(map);

	//chose driver according to tensor dimensionality N
	switch(N)	{
	case 2:
	{
		const int THREADS_PER_BLOCK = 512;
//		const int THREADS_PER_BLOCK = 2;

		uint2 dims = make_uint2(dimsa.get_dim(0), dimsa.get_dim(1) );
		//get b increments using the map
		uint2 b_incrs = make_uint2(dimsb.get_increment(map[1]), dimsb.get_increment(map[0]) );
		// setup execution parameters
		dim3 threads;
		// setup execution parameters
		threads.x = (dimsa.get_dim(1) < THREADS_PER_BLOCK) ? dimsa.get_dim(1) : THREADS_PER_BLOCK;
		dim3 grid(dimsa.get_dim(0));

		return new cuda_kern_copy_2d(pa, pb, threads, grid, b_incrs, dims, c, d);
	}
	case 4:
	{
		uint4 dims = make_uint4(dimsa.get_dim(3), dimsa.get_dim(2), dimsa.get_dim(1), dimsa.get_dim(0) );
		//get b increments using the map
		uint4 b_incrs = make_uint4(dimsb.get_increment(map[3]), dimsb.get_increment(map[2]), dimsb.get_increment(map[1]), dimsb.get_increment(map[0]) );

		// setup execution parameters
//		dim3 threads(dimsa.get_dim(3), dimsa.get_dim(2));
		//maximum possible block configuration is 32x32
		dim3 threads;
		threads.x = (dimsa.get_dim(3) > 32) ? 32 : dimsa.get_dim(3);
		threads.y = (dimsa.get_dim(2) > 32) ? 32 : dimsa.get_dim(2);
		dim3 grid(dimsa.get_dim(1), dimsa.get_dim(0));

		return new cuda_kern_copy_4d(pa, pb, threads, grid, b_incrs, dims, c, d);
	}
	case 6:
	{
		uint3 dims2 = make_uint3(dimsa.get_dim(5), dimsa.get_dim(4), dimsa.get_dim(3) );
		//get incriments using the map
		uint3 b_incrs1 = make_uint3(dimsb.get_increment(map[5]), dimsb.get_increment(map[4]), dimsb.get_increment(map[3]));
		uint3 b_incrs2 = make_uint3(dimsb.get_increment(map[2]), dimsb.get_increment(map[1]), dimsb.get_increment(map[0]));
//
//		// setup execution parameters
		//maximum possible block configuration is 8x8x16
//		dim3 threads(dimsa.get_dim(5), dimsa.get_dim(4), dimsa.get_dim(3));
		dim3 threads;
		threads.x = (dimsa.get_dim(5) > 16) ? 16 : dimsa.get_dim(5);
		threads.y = (dimsa.get_dim(4) > 8) ? 8 : dimsa.get_dim(4);
		threads.z = (dimsa.get_dim(3) > 8) ? 8 : dimsa.get_dim(3);
//		threads.x = (dimsa.get_dim(5) > 2) ? 2 : dimsa.get_dim(5);
//		threads.y = (dimsa.get_dim(4) > 2) ? 2 : dimsa.get_dim(4);
//		threads.z = (dimsa.get_dim(3) > 2) ? 2 : dimsa.get_dim(3);
		dim3 grid(dimsa.get_dim(2), dimsa.get_dim(1), dimsa.get_dim(0));

		return new cuda_kern_copy_6d(pa, pb, threads, grid, b_incrs1, b_incrs2, dims2, c, d);
	}
	//no default copy
	default:
		return NULL;
	}

}

template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<1> &dimsa,
		const permutation<1> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<2> &dimsa,
		const permutation<2> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<3> &dimsa,
		const permutation<3> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<4> &dimsa,
		const permutation<4> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<5> &dimsa,
		const permutation<5> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(cuda_pointer<const double> pa, cuda_pointer<double> pb, const dimensions<6> &dimsa,
		const permutation<6> &perma, const double &c, const double &d);

} // namespace libtensor
