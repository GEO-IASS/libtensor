#include "hip/hip_runtime.h"
#include "cuda_kern_copy_generic.h"
#include "cuda_kern_copy_2d.h"
#include "cuda_kern_copy_4d.h"
#include "cuda_kern_copy_6d.h"
#include <iostream>

namespace libtensor {

//template<size_t N>
const char *cuda_kern_copy_generic::k_clazz = "cuda_kern_copy_generic";

//generic implementation of copy operation doesn't do permutation!
//template<size_t N>
void cuda_kern_copy_generic::run() {

	//no default copy
	//do nothing

//	// setup execution parameters
////	dim3 threads(m_dimsa.get_dim(0));
//	dim3 threads(dim1);
//	dim3 grid;
//	if (N > 1)
//	{
////		grid.x = m_dimsa.get_dim(1);
//		grid.x = dim2;
//	} else {
//		grid.x = 1;
//	}
//	//run kernel
//
//	//kernel call
//	if (m_c == 1) {
//		cuda::generic_copy_tensor<<<grid, threads>>>(m_pa, m_pb, size);
//	} else {
//		cuda::generic_add_copy_tensor<<<grid, threads>>>(m_pa, m_pb, size, m_c);
//	}
}


template<size_t N>
cuda_kern_copy_generic *cuda_kern_copy_generic::match(const double *pa, double *pb, const dimensions<N> &dimsa,
		const permutation<N> &perma, const double &c, const double &d) {

	permutation<N> perma_i(perma, true);
	dimensions<N> dimsb(dimsa);
	dimsb.permute(perma);

	sequence<N, size_t>  map(0);
		for (size_t i = 0; i < N ; i++) {
		map[i] = i;
	}
	perma_i.apply(map);

	std::cout << "\n Map: " << map[0] << ", "<< map[1] << ", "<< map[2] << ", " << map[3] <<"\n";

	//chose driver according to tensor dimensionality N
	switch(N)	{
	case 2:
	{
		//get b increments using the map
		uint2 b_incrs = make_uint2(dimsb.get_increment(map[1]), dimsb.get_increment(map[0]) );
		std::cout << "dimsb dimensions: " << dimsb.get_dim(0) << ", " << dimsb.get_dim(1) << "\n";
		std::cout << "b_incrs: " << b_incrs.x << ", " << b_incrs.y << "\n " ;
		// setup execution parameters
		dim3 threads(dimsa.get_dim(1));
		dim3 grid(dimsa.get_dim(0));

		return new cuda_kern_copy_2d(pa, pb, threads, grid, b_incrs, c, d);
//		return NULL;
	}
	case 4:
	{
		//get b increments using the map
		uint4 b_incrs = make_uint4(dimsb.get_increment(map[3]), dimsb.get_increment(map[2]), dimsb.get_increment(map[1]), dimsb.get_increment(map[0]) );
		std::cout << "dimsb dimensions: " << dimsb.get_dim(0) << ", " << dimsb.get_dim(1) << ", " << dimsb.get_dim(2)  << ", " << dimsb.get_dim(3) << "\n";

		std::cout << "b_incrs: " << b_incrs.x << ", " << b_incrs.y << ", " << b_incrs.z << ", " << b_incrs.w << "\n " ;
		// setup execution parameters
		dim3 threads(dimsa.get_dim(3), dimsa.get_dim(2));
		dim3 grid(dimsa.get_dim(1), dimsa.get_dim(0));

		return new cuda_kern_copy_4d(pa, pb, threads, grid, b_incrs, c, d);
	}
	case 6:
	{
		//get incriments using the map
		uint3 b_incrs1 = make_uint3(dimsb.get_increment(map[5]), dimsb.get_increment(map[4]), dimsb.get_increment(map[3]));
		uint3 b_incrs2 = make_uint3(dimsb.get_increment(map[2]), dimsb.get_increment(map[1]), dimsb.get_increment(map[0]));
//
//		// setup execution parameters
		dim3 threads(dimsa.get_dim(5), dimsa.get_dim(4), dimsa.get_dim(3));
		dim3 grid(dimsa.get_dim(2), dimsa.get_dim(1), dimsa.get_dim(0));

		return new cuda_kern_copy_6d(pa, pb, threads, grid, b_incrs1, b_incrs2, c, d);
	}
	//no default copy
	default:
//		return new cuda_kern_copy_generic<N>(pa, pb, c);
		return NULL;
	}

//	cuda_kern_copy_generic<N> cuda_kern_generic(pa, pb, dimsa, c);
//	cuda_add_kern.m_dimsa = dimsa;
//	cuda_add_kern.m_c = c;

//	return this;

//	return new cuda_kern_copy_generic(cuda_kern_generic);
}

template cuda_kern_copy_generic *cuda_kern_copy_generic::match(const double *pa, double *pb, const dimensions<2> &dimsa,
		const permutation<2> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(const double *pa, double *pb, const dimensions<4> &dimsa,
		const permutation<4> &perma, const double &c, const double &d);
template cuda_kern_copy_generic *cuda_kern_copy_generic::match(const double *pa, double *pb, const dimensions<6> &dimsa,
		const permutation<6> &perma, const double &c, const double &d);

} // namespace libtensor
